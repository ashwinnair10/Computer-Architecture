#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void matrixadd(int *a,int *b, int *c,int n){
    int i=blockIdx.y*blockDim.y+threadIdx.y;
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    if(i<n&&j<n){
        c[n*n+j]=a[n*n+j]+b[n*n+j];
    }
}
int main(){
    int n=3;
    int a[3][3]={{1,1,1},{2,2,2},{3,3,3}};
    int b[3][3]={{-11,1,1},{2,2,2},{3,3,3}};
    int c[3][3]={{0}};
    int *da,*db,*dc;
    hipMalloc((void**)&da,n*n*sizeof(int));
    hipMalloc((void**)&db,n*n*sizeof(int));
    hipMalloc((void**)&dc,n*n*sizeof(int));
    hipMemcpy(da,a,n*n*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(db,b,n*n*sizeof(int),hipMemcpyDeviceToHost);
    dim3 blockSize(16,16);
    dim3 gridSize((n+blockSize.x-1)/blockSize.x,(n+blockSize.y-1)/blockSize.y);
    matrixadd<<<gridSize,blockSize>>>(da,db,dc,n);
    hipMemcpy(c,dc,n*n*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<c[i][j]<<' ';
        }
        cout<<'\n';
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}