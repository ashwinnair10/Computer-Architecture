#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int* a, int* b, int* c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 5;
    int a[n] = {1, 1, 1, 1, 1};
    int b[n] = {3, 4, 5, 4, 3};
    int c[n] = {0};
    int *da, *db, *dc;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc((void**)&da, n * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipMalloc((void**)&db, n * sizeof(int));
    if (err != hipSuccess) {
        hipFree(da);
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipMalloc((void**)&dc, n * sizeof(int));
    if (err != hipSuccess) {
        hipFree(da);
        hipFree(db);
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy data to device
    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch
    int tpb = 256;
    int bpg = (n + tpb - 1) / tpb;
    add<<<bpg, tpb>>>(da, db, dc, n);

    // Copy result back to host
    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < n; i++) {
        std::cout << c[i] << ' ';
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}
